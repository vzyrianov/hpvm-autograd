#include "hip/hip_runtime.h"
//===--------------------------- approxtechniques.cu ---------------------===//
//
//===----------------------------------------------------------------------===//
//
//  This file  consists of our CUDA-based implementation for convolution approximations
//
//  *Supported Approximations: Perforated Convolutions, Filter Sampling
//
//  FP32 Convolution Routine:  `tensorConvApprox`
//  FP16 Convolution Routine:  `tensorConvApproxHalf2`
// 
//  NOTE: These approximations are tuned for NVIDIA Jetson Tx2 device
//
//  Author: Akash Kothari
//===----------------------------------------------------------------------===//

#include "tensor_utils.h"
#include "approx_utils.h"
#include "debug.h"
#include "global_data.h"
#include "fp16_gemm.h"
#include "fp16_conversion.h"
#include "profiling.h"

extern "C" {

__global__ void convToGemm(float *const __restrict__ output,
                           const float *const __restrict input, const int N,
                           const int C, const int H, const int W, const int KH,
                           const int KW, const int V_pad, const int H_pad,
                           const int H_out, const int W_out, const int V_stride,
                           const int H_stride, const int num_filter_elem) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  if (n < N) {
    const int c =
        tx % (C * H_out * W_out) / (H_out * W_out); // output chan number
    const int h =
        tx % (H_out * W_out) / W_out; // output height index (row number)
    const int w = tx % W_out;         // output width index (col number)
    const int inH = h * V_stride - V_pad;
    const int inW = w * H_stride - H_pad;
    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            (c * KH + i) * KW + j; // index of this filter element
        const int out_index =
            ((n * C * KH * KW + filter_elem_num) * H_out + h) * W_out + w;
        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
          output[out_index] =
              input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        else
          output[out_index] = 0;
      }
    }
  }
}

__global__ void convToGemmFullInput(
    float *const __restrict__ output, const float *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride,
    const int skip_every, const int skip_offset) {
  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  const int c = tx % (C * H_out * W_out) / (H_out * W_out); // output chan
                                                            // number
  const int h =
      tx % (H_out * W_out) / W_out;     // output height index (row number)_
  const int w = tx % W_out;             // output width index (col number)
  const int inH = h * V_stride - V_pad; // input height index (row number)
  const int inW = w * H_stride - H_pad; // input width index (col number)
  if (n < N) {                          // is thread id within bounds?
    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            (c * KH + i) * KW + j; // index of this filter elemen
        if (filter_elem_num % skip_every != skip_every - 1 - skip_offset) {
          int output_col =
              filter_elem_num - ((filter_elem_num + skip_every) / skip_every);
          if (skip_every == 1)
            output_col = filter_elem_num;
          if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
            output[((output_col * N + n) * H_out + h) * W_out + w] =
                input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
          else
            output[((output_col * N + n) * H_out + h) * W_out + w] = 0;
        }
      }
    }
  }
}

__global__ void
convToGemmHalfInputNew(__half *const __restrict__ output,
                       const __half *const __restrict input, const int N,
                       const int C, const int H, const int W, const int KH,
                       const int KW, const int V_pad, const int H_pad,
                       const int H_out, const int W_out, const int V_stride,
                       const int H_stride, const int reduced_filter_elem,
                       const int skip_every, const int skip_offset) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  const int c = tx % (C * H_out * W_out) / (H_out * W_out); // output chan
                                                            // number
  const int h = tx % (H_out * W_out) / W_out; // output height index (row
                                              // number)
  const int w = tx % W_out;                   // output width index (col number)
  const int inH = h * V_stride - V_pad;       // input height index (row number)
  const int inW = w * H_stride - H_pad;       // input width index (col number)
  if (n < N) {                                // is thread id within bounds?
    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            (c * KH + i) * KW + j; // index of this filter element
        if (filter_elem_num % skip_every != skip_offset) {
          int output_col =
              filter_elem_num - (filter_elem_num / skip_every +
                                 (filter_elem_num % skip_every > skip_offset));
          if (skip_every == 1)
            output_col = filter_elem_num;
          if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
            output[((output_col * N + n) * H_out + h) * W_out + w] =
                input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
          else
            output[((output_col * N + n) * H_out + h) * W_out + w] = 0;
        }
      }
    }
  }
}

__global__ void convToGemmHalf(__half *const __restrict__ output,
                               const __half *const __restrict input,
                               const int N, const int C, const int H,
                               const int W, const int KH, const int KW,
                               const int V_pad, const int H_pad,
                               const int H_out, const int W_out,
                               const int V_stride, const int H_stride) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread i
  const int n = tx / (C * H_out * W_out);               // output image numbe
  const int c = tx % (C * H_out * W_out) / (H_out * W_out); // output chan numbe
  const int h = tx % (H_out * W_out) / W_out; // output height index (row number
  const int w = tx % W_out;                   // output width index (col number
  const int inH = h * V_stride - V_pad;
  const int inW = w * H_stride - H_pad; // input width index (col number)
  if (n < N) {                          // is thread id within bounds?
    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            (c * KH + i) * KW + j; // index of this filter element
        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W) {
          output[((filter_elem_num * N + n) * H_out + h) * W_out + w] =
              input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        } else {
          output[((filter_elem_num * N + n) * H_out + h) * W_out + w] = 0;
        }
      }
    }
  }
}

__global__ void convToGemmHalfInputNewIrregular(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride,
    const int reduced_filter_elem, const int skip_every,
    const int skip_offset) {
  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  const int c = tx % (C * H_out * W_out) / (H_out * W_out); // output chan
                                                            // number
  const int h = tx % (H_out * W_out) / W_out; // output height index (row
                                              // number)
  const int w = tx % W_out;                   // output width index (col number)
  const int inH = h * V_stride - V_pad;       // input height index (row number)
  const int inW = w * H_stride - H_pad;       // input width index (col number)
  if (n < N) {                                // is thread id within bounds?
    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {

        const int filter_elem_num =
            (c * KH + i) * KW + j; // index of this filter element
        if ((filter_elem_num - skip_offset) % skip_every) {
          const int condition = (filter_elem_num < skip_offset);
          const int output_col =
              condition * filter_elem_num +
              (!condition) *
                  (filter_elem_num -
                   ((filter_elem_num + 1 - skip_offset) / skip_every) -
                   ((filter_elem_num + 1 - skip_offset) % skip_every > 0));
          const int out_index =
              ((n * reduced_filter_elem + output_col) * H_out + h) * W_out + w;
          //((output_col*N + n) * H_out + h) * W_out + w;
          if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
            output[out_index] =
                input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
          else
            output[out_index] = 0;
        }
      }
    }
  }
}

__global__ void convToGemmHalfInputNewIrregular2(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride,
    const int reduced_filter_elem, const int skip_every,
    const int skip_offset) {
  
  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  const int c = tx % (C * H_out * W_out) / (H_out * W_out); // output chan
                                                            // number
  const int h = tx % (H_out * W_out) / W_out; // output height index (row
                                              // number)
  const int w = tx % W_out;                   // output width index (col number)
  const int inH = h * V_stride - V_pad;       // input height index (row number)
  const int inW = w * H_stride - H_pad;       // input width index (col number)
  if (n < N) {                                // is thread id within bounds?
    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {

        const int filter_elem_num =
            (c * KH + i) * KW + j; // index of this filter element
        if ((filter_elem_num - skip_offset) % skip_every) {
          const int condition = (filter_elem_num < skip_offset);
          const int output_col =
              condition * filter_elem_num +
              (!condition) *
                  (filter_elem_num -
                   ((filter_elem_num + 1 - skip_offset) / skip_every) -
                   ((filter_elem_num + 1 - skip_offset) % skip_every > 0));

          const int out_index = ((output_col * N + n) * H_out + h) * W_out + w;

          if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
            output[out_index] =
                input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
          else
            output[out_index] = 0;
        }
      }
    }
  }
}

__global__ void convToGemmHalf2(__half *const __restrict__ output,
                                const __half *const __restrict input,
                                const int N, const int C, const int H,
                                const int W, const int KH, const int KW,
                                const int V_pad, const int H_pad,
                                const int H_out, const int W_out,
                                const int V_stride, const int H_stride,
                                const int num_filter_elem) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  if (n < N) {
    const int c =
        tx % (C * H_out * W_out) / (H_out * W_out); // output chan number
    const int h =
        tx % (H_out * W_out) / W_out; // output height index (row number)
    const int w = tx % W_out;         // output width index (col number)
    const int inH = h * V_stride - V_pad;
    const int inW = w * H_stride - H_pad;
    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            (c * KH + i) * KW + j; // index of this filter element
        const int out_index =
            ((n * C * KH * KW + filter_elem_num) * H_out + h) * W_out + w;
        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
          output[out_index] =
              input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        else
          output[out_index] = 0;
      }
    }
  }
}

__global__ void
convToGemmPerfRow(float *const __restrict__ output,
                  const float *const __restrict input, const int N, const int C,
                  const int H, const int W, const int KH, const int KW,
                  const int V_pad, const int H_pad, const int H_out,
                  const int W_out, const int V_stride, const int H_stride,
                  const int x, const int start, const int H_eff) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_eff * W_out);               // output image number
  if (n < N) {
    const int c =
        tx % (C * H_eff * W_out) / (H_eff * W_out); // output chan number
    const int h =
        tx % (H_eff * W_out) / W_out; // output height index (row number)
    const int w = tx % W_out;         // output width index (col number)
    int h_index;
    if (h < start) {
      h_index = h;
    } else {
      h_index = ((h - start + 1) * x) / (x - 1) +
                (((h - start + 1) * x) % (x - 1) > 0) + start - 1;
    }
    const int inH = h_index * V_stride - V_pad;
    const int inW = w * H_stride - H_pad; // input width index (col number)

    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            c * KH * KW + i * KW + j; // index of this filter element
        const int out_index =
            ((n * C * KH * KW + filter_elem_num) * H_eff + h) * W_out + w;

        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
          output[out_index] =
              input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        else
          output[out_index] = 0;
      }
    }
  }
}

__global__ void approxInterpolateRow(int N, int old_h, int j, int c, int h,
                                     int w, float *old_data, float *new_data,
                                     int x, int start) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (c * h * w);                       // output image number
  if (n < N) {
    const int ch = tx % (c * h * w) / (h * w); // filter number
    const int row = tx % (h * w) / w; // output height index (row number)
    const int col = tx % w;           // output width index (col number)

    if (row < start) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * old_h * w) + ch * (old_h * w) + row * (w) + col];
    } else if (row == h - 1) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * old_h * w) + ch * (old_h * w) + (old_h - 1) * (w) +
                   col];
    } else if (row == 0) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * old_h * w) + ch * (old_h * w) + 0 * (w) + col];
    } else if ((row - start) % x == 0) {
      int row_index = row - ((row + 1 - start) / x);
      int output_index =
          n * (c * old_h * w) + ch * (old_h * w) + row_index * (w) + col;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          (old_data[output_index] + old_data[output_index - w]) / 2;
    } else {
      int row_index =
          row - ((row + 1 - start) / x) - ((row + 1 - start) % x > 0);
      int output_index =
          n * (c * old_h * w) + ch * (old_h * w) + row_index * (w) + col;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[output_index];
    }
  }
}

__global__ void
convToGemmPerfCol(float *const __restrict__ output,
                  const float *const __restrict input, const int N, const int C,
                  const int H, const int W, const int KH, const int KW,
                  const int V_pad, const int H_pad, const int H_out,
                  const int W_out, const int V_stride, const int H_stride,
                  const int x, const int start, const int W_eff) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_eff);               // output image number
  if (n < N) {
    const int c =
        tx % (C * H_out * W_eff) / (H_out * W_eff); // output chan number
    const int h =
        tx % (H_out * W_eff) / W_eff; // output height index (row number)
    const int w = tx % W_eff;         // output width index (col number)
    int w_index;
    if (w < start) {
      w_index = w;
    } else {
      w_index = ((w - start + 1) * x) / (x - 1) +
                (((w - start + 1) * x) % (x - 1) > 0) + start - 1;
    }
    const int inW = w_index * H_stride - H_pad;
    const int inH = h * V_stride - V_pad; // input height index (row number)

    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            c * KH * KW + i * KW + j; // index of this filter element
        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
          output[((n * C * KH * KW + filter_elem_num) * H_out + h) * W_eff +
                 w] = input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        else
          output[((n * C * KH * KW + filter_elem_num) * H_out + h) * W_eff +
                 w] = 0;
      }
    }
  }
}

__global__ void approxInterpolateCol(int N, int old_w, int b, int c, int h,
                                     int w, float *old_data, float *new_data,
                                     int x, int start) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (c * h * w);                       // output image number
  if (n < N) {
    const int ch = tx % (c * h * w) / (h * w); // output chan number
    const int row = tx % (h * w) / w; // output height index (row number)
    const int col = tx % w;           // output width index (col number)

    if (col < start) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * h * old_w) + ch * (h * old_w) + row * old_w + col];
    } else if (col == w - 1) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * h * old_w) + ch * (h * old_w) + row * (old_w) +
                   old_w - 1];
    } else if (col == 0) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * h * old_w) + ch * (h * old_w) + row * (old_w)];
    } else if ((col - start) % x == 0) {
      int col_index = col - ((col + 1 - start) / x);
      int output_index =
          n * (c * h * old_w) + ch * (h * old_w) + row * old_w + col_index;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          (old_data[output_index] + old_data[output_index - 1]) / 2;
    } else {
      int col_index =
          col - ((col + 1 - start) / x) - ((col + 1 - start) % x > 0);
      int output_index =
          n * (c * h * old_w) + ch * (h * old_w) + row * old_w + col_index;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[output_index];
    }
  }
}

__global__ void convToGemmPerfRowHalf(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride, const int x,
    const int start, const int H_eff) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_eff * W_out);               // output image number
  if (n < N) {
    const int c =
        tx % (C * H_eff * W_out) / (H_eff * W_out); // output chan number
    const int h =
        tx % (H_eff * W_out) / W_out; // output height index (row number)
    const int w = tx % W_out;         // output width index (col number)
    int h_index;
    if (h < start) {
      h_index = h;
    } else {
      h_index = ((h - start + 1) * x) / (x - 1) +
                (((h - start + 1) * x) % (x - 1) > 0) + start - 1;
    }
    const int inH = h_index * V_stride - V_pad;
    const int inW = w * H_stride - H_pad; // input width index (col number)

    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            c * KH * KW + i * KW + j; // index of this filter element
        const int out_index =
            ((n * C * KH * KW + filter_elem_num) * H_eff + h) * W_out + w;
        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
          output[out_index] =
              input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        else
          output[out_index] = 0;
      }
    }
  }
}

__global__ void convToGemmPerfRowHalf2(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride, const int x,
    const int start, const int H_eff) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_eff * W_out);               // output image numbe
  if (n < N) {
    const int c =
        tx % (C * H_eff * W_out) / (H_eff * W_out); // output chan number
    const int h =
        tx % (H_eff * W_out) / W_out; // output height index (row number)
    const int w = tx % W_out;         // output width index (col number)
    int h_index;
    if (h < start) {
      h_index = h;
    } else {
      h_index = ((h - start + 1) * x) / (x - 1) +
                (((h - start + 1) * x) % (x - 1) > 0) + start - 1;
    }
    const int inH = h_index * V_stride - V_pad;
    const int inW = w * H_stride - H_pad; // input width index (col number)

    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            c * KH * KW + i * KW + j; // index of this filter element
        const int out_index =
            ((filter_elem_num * N + n) * H_eff + h) * W_out + w;

        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
          output[out_index] =
              input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        else
          output[out_index] = 0;
      }
    }
  }
}

__global__ void approxInterpolateRowHalf(int N, int old_h, int j, int c, int h,
                                         int w, __half *old_data,
                                         __half *new_data, int x, int start) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (c * h * w);                       // output image number
  if (n < N) {

    const int ch = tx % (c * h * w) / (h * w); // filter number
    const int row = tx % (h * w) / w; // output height index (row number)
    const int col = tx % w;           // output width index (col number)

    if (row < start) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * old_h * w) + ch * (old_h * w) + row * (w) + col];
    } else if (row == h - 1) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * old_h * w) + ch * (old_h * w) + (old_h - 1) * (w) +
                   col];
    } else if (row == 0) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * old_h * w) + ch * (old_h * w) + 0 * (w) + col];
    } else if ((row - start) % x == 0) {
      int row_index = row - ((row + 1 - start) / x);
      int output_index =
          n * (c * old_h * w) + ch * (old_h * w) + row_index * (w) + col;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          __hdiv(__hadd(old_data[output_index], old_data[output_index - w]), 2);
    } else {
      int row_index =
          row - ((row + 1 - start) / x) - ((row + 1 - start) % x > 0);
      int output_index =
          n * (c * old_h * w) + ch * (old_h * w) + row_index * (w) + col;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[output_index];
    }
  }
}

__global__ void approxInterpolateRowHalf2(int N, int old_h, int b, int c, int h,
                                          int w, __half *old_data,
                                          __half *new_data, int x, int start) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (c * h * w);                       // output image number
  if (n < N) {

    const int ch = tx % (c * h * w) / (h * w); // filter number
    const int row = tx % (h * w) / w; // output height index (row number)
    const int col = tx % w;           // output width index (col number
    if (row < start) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[ch * (b * old_h * w) + n * (old_h * w) + row * (w) + col];
    } else if (row == h - 1) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[ch * (b * old_h * w) + n * (old_h * w) + (old_h - 1) * (w) +
                   col];
    } else if (row == 0) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[ch * (b * old_h * w) + n * (old_h * w) + 0 * (w) + col];
    } else if ((row - start) % x == 0) {
      const int row_index = row - ((row + 1 - start) / x);
      const int output_index =
          ch * (b * old_h * w) + n * (old_h * w) + row_index * (w) + col;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          __hdiv(__hadd(old_data[output_index], old_data[output_index - w]), 2);
    } else {
      const int row_index =
          row - ((row + 1 - start) / x) - ((row + 1 - start) % x > 0);
      const int output_index =
          ch * (b * old_h * w) + n * (old_h * w) + row_index * (w) + col;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[output_index];
    }
  }
}

__global__ void convToGemmPerfColHalf(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride, const int x,
    const int start, const int W_eff) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_eff);               // output image number
  if (n < N) {
    const int c =
        tx % (C * H_out * W_eff) / (H_out * W_eff); // output chan number
    const int h =
        tx % (H_out * W_eff) / W_eff; // output height index (row number)
    const int w = tx % W_eff;         // output width index (col number)
    int w_index;
    if (w < start) {
      w_index = w;
    } else {
      w_index = ((w - start + 1) * x) / (x - 1) +
                (((w - start + 1) * x) % (x - 1) > 0) + start - 1;
    }
    const int inW = w_index * H_stride - H_pad;
    const int inH = h * V_stride - V_pad; // input height index (row number)

    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            c * KH * KW + i * KW + j; // index of this filter element
        const int out_index =
            ((n * C * KH * KW + filter_elem_num) * H_out + h) * W_eff + w;
        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
          output[out_index] =
              input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        else
          output[out_index] = 0;
      }
    }
  }
}

__global__ void convToGemmPerfColHalf2(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride, const int x,
    const int start, const int W_eff) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_eff);               // output image number
  if (n < N) {
    const int c =
        tx % (C * H_out * W_eff) / (H_out * W_eff); // output chan number
    const int h =
        tx % (H_out * W_eff) / W_eff; // output height index (row number)
    const int w = tx % W_eff;         // output width index (col number)
    int w_index;
    if (w < start) {
      w_index = w;
    } else {
      w_index = ((w - start + 1) * x) / (x - 1) +
                (((w - start + 1) * x) % (x - 1) > 0) + start - 1;
    }
    const int inW = w_index * H_stride - H_pad;
    const int inH = h * V_stride - V_pad; // input height index (row number)

    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            c * KH * KW + i * KW + j; // index of this filter elemen
        const int out_index =
            ((filter_elem_num * N + n) * H_out + h) * W_eff + w;
        if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
          output[out_index] =
              input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
        else
          output[out_index] = 0;
      }
    }
  }
}

__global__ void approxInterpolateColHalf(int N, int old_w, int b, int c, int h,
                                         int w, __half *old_data,
                                         __half *new_data, int x, int start) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (c * h * w);                       // output image number
  if (n < N) {
    const int ch = tx % (c * h * w) / (h * w); // output chan number
    const int row = tx % (h * w) / w; // output height index (row number)
    const int col = tx % w;           // output width index (col number)

    if (col < start) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * h * old_w) + ch * (h * old_w) + row * old_w + col];
    } else if (col == w - 1) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * h * old_w) + ch * (h * old_w) + row * (old_w) +
                   old_w - 1];
    } else if (col == 0) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[n * (c * h * old_w) + ch * (h * old_w) + row * (old_w)];
    } else if ((col - start) % x == 0) {
      int col_index = col - ((col + 1 - start) / x);
      int output_index =
          n * (c * h * old_w) + ch * (h * old_w) + row * old_w + col_index;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          __hdiv(__hadd(old_data[output_index], old_data[output_index - 1]), 2);
    } else {
      int col_index =
          col - ((col + 1 - start) / x) - ((col + 1 - start) % x > 0);
      int output_index =
          n * (c * h * old_w) + ch * (h * old_w) + row * old_w + col_index;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[output_index];
    }
  }
}

__global__ void approxInterpolateColHalf2(int N, int old_w, int b, int c, int h,
                                          int w, __half *old_data,
                                          __half *new_data, int x, int start) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (c * h * w);                       // output image number
  if (n < N) {
    const int ch = tx % (c * h * w) / (h * w); // output chan number
    const int row = tx % (h * w) / w; // output height index (row number)
    const int col = tx % w;           // output width index (col number)
    if (col < start) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[ch * (b * h * old_w) + n * (h * old_w) + row * old_w + col];

    } else if (col == w - 1) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[ch * (b * h * old_w) + n * (h * old_w) + row * (old_w) +
                   old_w - 1];

    } else if (col == 0) {
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[ch * (b * h * old_w) + n * (h * old_w) + row * (old_w)];

    } else if ((col - start) % x == 0) {
      const int col_index = col - ((col + 1 - start) / x);
      const int output_index =
          ch * (b * h * old_w) + n * (h * old_w) + row * old_w + col_index;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          __hdiv(__hadd(old_data[output_index], old_data[output_index - 1]), 2);
    } else {
      const int col_index =
          col - ((col + 1 - start) / x) - ((col + 1 - start) % x > 0);
      const int output_index =
          ch * (b * h * old_w) + n * (h * old_w) + row * old_w + col_index;
      new_data[n * (c * h * w) + ch * (h * w) + row * (w) + col] =
          old_data[output_index];
    }
  }
}

__global__ void
convToGemmFullInputRegular(float *const __restrict__ output,
                           const float *const __restrict input, const int N,
                           const int C, const int H, const int W, const int KH,
                           const int KW, const int V_pad, const int H_pad,
                           const int H_out, const int W_out, const int V_stride,
                           const int H_stride, const int reduced_filter_elem,
                           const int skip_every, const int skip_offset) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (H_out * W_out);                   // output image number
  if (n < N) {
    const int h =
        tx % (H_out * W_out) / W_out;     // output height index (row number)
    const int w = tx % W_out;             // output width index (col number)
    const int inH = h * V_stride - V_pad; // input height index (row number)
    const int inW = w * H_stride - H_pad; // input width index (col number)

#pragma unroll
    for (int fi = 0; fi < reduced_filter_elem; fi++) {
      const int ch = (fi * C) / reduced_filter_elem;
      const int offset = (skip_offset + ch) % skip_every;
      int in_index;
      if (fi < offset) {
        in_index = fi;
      } else {
        in_index = ((fi - offset + 1) * skip_every) / (skip_every - 1) +
                   (((fi - offset + 1) * skip_every) % (skip_every - 1) > 0) +
                   offset - 1;
      }

      const int i = (in_index % (KW * KH)) / KW;
      const int j = in_index % KW;
      const int out_index =
          ((n * reduced_filter_elem + fi) * H_out + h) * W_out + w;
      if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W) {
        output[out_index] =
            input[((n * C + ch) * H + (inH + i)) * W + (inW + j)];
      } else {
        output[out_index] = 0;
      }
    }
  }
}

__global__ void convToGemmFullInputIrregular(
    float *const __restrict__ output, const float *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride,
    const int reduced_filter_elem, const int skip_every,
    const int skip_offset) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (H_out * W_out);                   // output image number
  if (n < N) {
    const int h =
        tx % (H_out * W_out) / W_out;     // output height index (row number)
    const int w = tx % W_out;             // output width index (col number)
    const int inH = h * V_stride - V_pad; // input height index (row number)
    const int inW = w * H_stride - H_pad; // input width index (col number)

#pragma unroll
    for (int fi = 0; fi < reduced_filter_elem; fi++) {
      int in_index;
      if (fi < skip_offset) {
        in_index = fi;
      } else {
        in_index =
            ((fi - skip_offset + 1) * skip_every) / (skip_every - 1) +
            (((fi - skip_offset + 1) * skip_every) % (skip_every - 1) > 0) +
            skip_offset - 1;
      }
      const int ch = in_index / (KW * KH);
      const int i = (in_index % (KW * KH)) / KW;
      const int j = in_index % KW;
      const int out_index =
          ((n * reduced_filter_elem + fi) * H_out + h) * W_out + w;
      if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W) {
        output[out_index] =
            input[((n * C + ch) * H + (inH + i)) * W + (inW + j)];
      } else {
        output[out_index] = 0;
      }
    }
  }
}

__global__ void createReducedFiltersFullRegular(
    float *output, const float *const __restrict input, const int NF,
    const int num_filter_elem, const int reduced_filter_elem,
    const int channels, const int skip_every, const int skip_offset,
    const float fac) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int fIdx = tx / reduced_filter_elem;            // filter index
  if (fIdx < NF) {
    const int offset = tx % reduced_filter_elem; // offset within filter
    const int ch = (offset * channels) / reduced_filter_elem;
    const int channel_offset = (skip_offset + ch) % skip_every;
    int in_index;
    if (offset < channel_offset) {
      in_index = offset;
    } else {
      in_index =
          ((offset - channel_offset + 1) * skip_every) / (skip_every - 1) +
          (((offset - channel_offset + 1) * skip_every) % (skip_every - 1) >
           0) +
          channel_offset - 1;
    }

    output[fIdx * reduced_filter_elem + offset] =
        fac * input[num_filter_elem * fIdx + in_index];
  }
}

__global__ void createReducedFiltersFullIrregular(
    float *output, const float *const __restrict input, const int NF,
    const int num_filter_elem, const int reduced_filter_elem,
    const int skip_every, const int skip_offset, const float fac) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int fIdx = tx / reduced_filter_elem;            // filter index
  if (fIdx < NF) {
    const int offset = tx % reduced_filter_elem; // offset within filter
    int in_index;
    if (offset < skip_offset) {
      in_index = offset;
    } else {
      in_index =
          ((offset - skip_offset + 1) * skip_every) / (skip_every - 1) +
          (((offset - skip_offset + 1) * skip_every) % (skip_every - 1) > 0) +
          skip_offset - 1;
    }
    output[fIdx * reduced_filter_elem + offset] =
        fac * input[num_filter_elem * fIdx + in_index];
  }
}

__global__ void
convToGemmHalfInputRegular(__half *const __restrict__ output,
                           const __half *const __restrict input, const int N,
                           const int C, const int H, const int W, const int KH,
                           const int KW, const int V_pad, const int H_pad,
                           const int H_out, const int W_out, const int V_stride,
                           const int H_stride, const int reduced_filter_elem,
                           const int skip_every, const int skip_offset) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  if (n < N) {
    const int ch =
        tx % (C * H_out * W_out) / (H_out * W_out); // output chan number
    const int h =
        tx % (H_out * W_out) / W_out;     // output height index (row number)
    const int w = tx % W_out;             // output width index (col number)
    const int inH = h * V_stride - V_pad; // input height index (row number)
    const int inW = w * H_stride - H_pad; // input width index (col number)

#pragma unroll
    for (int ki = 0; ki < reduced_filter_elem / C; ki++) {
      const int fi = ch * (reduced_filter_elem / C) + ki;
      const int offset = (skip_offset + ch) % skip_every;

      const bool condition = (fi < offset);
      const int in_index =
          condition * fi +
          (!condition) *
              (((fi - offset + 1) * skip_every) / (skip_every - 1) +
               (((fi - offset + 1) * skip_every) % (skip_every - 1) > 0) +
               offset - 1);

      const int i = (in_index % (KW * KH)) / KW;
      const int j = in_index % KW;
      const int out_index =
          ((n * reduced_filter_elem + fi) * H_out + h) * W_out + w;
      if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W) {
        output[out_index] =
            input[((n * C + ch) * H + (inH + i)) * W + (inW + j)];
      } else {
        output[out_index] = 0;
      }
    }
  }
}

__global__ void convToGemmHalfInputRegular2(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride,
    const int reduced_filter_elem, const int skip_every,
    const int skip_offset) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  if (n < N) {
    const int ch =
        tx % (C * H_out * W_out) / (H_out * W_out); // output chan number
    const int h =
        tx % (H_out * W_out) / W_out;     // output height index (row number)
    const int w = tx % W_out;             // output width index (col number)
    const int inH = h * V_stride - V_pad; // input height index (row number)
    const int inW = w * H_stride - H_pad; // input width index (col number)

#pragma unroll
    for (int ki = 0; ki < reduced_filter_elem / C; ki++) {

      const int fi = ch * (reduced_filter_elem / C) + ki;
      const int offset = (skip_offset + ch) % skip_every;
      const int condition = (fi < offset);
      const int in_index =
          condition * fi +
          (!condition) *
              (((fi - offset + 1) * skip_every) / (skip_every - 1) +
               (((fi - offset + 1) * skip_every) % (skip_every - 1) > 0) +
               offset - 1);

      const int i = (in_index % (KW * KH)) / KW;
      const int j = in_index % KW;
      const int out_index = ((fi * N + n) * H_out + h) * W_out + w;
      if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W) {
        output[out_index] =
            input[((n * C + ch) * H + (inH + i)) * W + (inW + j)];
      } else {
        output[out_index] = 0;
      }
    }
  }
}

__global__ void convToGemmHalfInputIrregular(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride,
    const int reduced_filter_elem, const int skip_every,
    const int skip_offset) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (H_out * W_out);                   // output image number
  if (n < N) {
    const int h =
        tx % (H_out * W_out) / W_out;     // output height index (row number)
    const int w = tx % W_out;             // output width index (col number)
    const int inH = h * V_stride - V_pad; // input height index (row number)
    const int inW = w * H_stride - H_pad; // input width index (col number)

#pragma unroll
    for (int fi = 0; fi < reduced_filter_elem; fi++) {
      const int condition = (fi < skip_offset);
      const int in_index =
          condition * fi +
          (!condition) *
              (((fi - skip_offset + 1) * skip_every) / (skip_every - 1) +
               (((fi - skip_offset + 1) * skip_every) % (skip_every - 1) > 0) +
               skip_offset - 1);

      const int ch = in_index / (KW * KH);
      const int i = (in_index % (KW * KH)) / KW;
      const int j = in_index % KW;
      const int out_index =
          ((n * reduced_filter_elem + fi) * H_out + h) * W_out + w;
      if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W) {
        output[out_index] =
            input[((n * C + ch) * H + (inH + i)) * W + (inW + j)];
      } else {
        output[out_index] = 0;
      }
    }
  }
}

__global__ void convToGemmHalfInputIrregular2(
    __half *const __restrict__ output, const __half *const __restrict input,
    const int N, const int C, const int H, const int W, const int KH,
    const int KW, const int V_pad, const int H_pad, const int H_out,
    const int W_out, const int V_stride, const int H_stride,
    const int reduced_filter_elem, const int skip_every,
    const int skip_offset) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (H_out * W_out);                   // output image number
  if (n < N) {
    const int h =
        tx % (H_out * W_out) / W_out;     // output height index (row number)
    const int w = tx % W_out;             // output width index (col number)
    const int inH = h * V_stride - V_pad; // input height index (row number)
    const int inW = w * H_stride - H_pad; // input width index (col number)
#pragma unroll
    for (int fi = 0; fi < reduced_filter_elem; fi++) {
      const int condition = (fi < skip_offset);
      const int in_index =
          condition * fi +
          (!condition) *
              (((fi - skip_offset + 1) * skip_every) / (skip_every - 1) +
               (((fi - skip_offset + 1) * skip_every) % (skip_every - 1) > 0) +
               skip_offset - 1);

      const int ch = in_index / (KW * KH);
      const int i = (in_index % (KW * KH)) / KW;
      const int j = in_index % KW;
      const int out_index = ((fi * N + n) * H_out + h) * W_out + w;
      if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W) {
        output[out_index] =
            input[((n * C + ch) * H + (inH + i)) * W + (inW + j)];
      } else {
        output[out_index] = 0;
      }
    }
  }
}

__global__ void createReducedFiltersHalfRegular(
    __half *output, const __half *const __restrict input, const int NF,
    const int num_filter_elem, const int reduced_filter_elem,
    const int channels, const int skip_every, const int skip_offset,
    const float fac) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id

  const int fIdx = tx / reduced_filter_elem; // filter index
  if (fIdx < NF) {
    const int offset = tx % reduced_filter_elem; // offset within filter
    const int ch = (offset * channels) / reduced_filter_elem;
    const int channel_offset = (skip_offset + ch) % skip_every;
    const int condition = (offset < channel_offset);
    const int in_index =
        condition * offset +
        (!condition) *
            (((offset - channel_offset + 1) * skip_every) / (skip_every - 1) +
             (((offset - channel_offset + 1) * skip_every) % (skip_every - 1) >
              0) +
             channel_offset - 1);

    output[fIdx * reduced_filter_elem + offset] =
        __hmul(__float2half_rn(fac), input[num_filter_elem * fIdx + in_index]);
  }
}

__global__ void createReducedFiltersHalfIrregular(
    __half *output, const __half *const __restrict input, const int NF,
    const int num_filter_elem, const int reduced_filter_elem,
    const int skip_every, const int skip_offset, const float fac) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int fIdx = tx / reduced_filter_elem;            // filter index

  if (fIdx < NF) {

    const int offset = tx % reduced_filter_elem; // offset within filter
    const int condition = (offset < skip_offset);

    int in_index =
        condition * offset +
        (!condition) *
            (((offset - skip_offset + 1) * skip_every) / (skip_every - 1) +
             (((offset - skip_offset + 1) * skip_every) % (skip_every - 1) >
              0) +
             skip_offset - 1);

    output[fIdx * reduced_filter_elem + offset] =
        __hmul(__float2half_rn(fac), input[num_filter_elem * fIdx + in_index]);
  }
}

// produces N COL MAJOR matrixes with H_out*W_out rows and reduced_filter_elem
// cols
__global__ void
convToGemmApprox(float *const __restrict__ output,
                 const float *const __restrict input, const int N, const int C,
                 const int H, const int W, const int KH, const int KW,
                 const int V_pad, const int H_pad, const int H_out,
                 const int W_out, const int V_stride, const int H_stride,
                 const int reduced_filter_elem, const int skip_every) {

  const int tx = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int n = tx / (C * H_out * W_out);               // output image number
  const int c = tx % (C * H_out * W_out) / (H_out * W_out); // output chan
                                                            // number
  const int h = tx % (H_out * W_out) / W_out; // output height index (row
                                              // number)
  const int w = tx % W_out;                   // output width index (col number)
  const int inH = h * V_stride - V_pad;       // input height index (row number)
  const int inW = w * H_stride - H_pad;       // input width index (col number)
  if (n < N) {                                // is thread id within bounds?
    for (int i = 0; i < KH; i++) {
      for (int j = 0; j < KW; j++) {
        const int filter_elem_num =
            (c * KH + i) * KW + j; // index of this filter element
        if (filter_elem_num % skip_every !=
            skip_every - 1) { // are we including this filter element?
          const int output_col =
              filter_elem_num -
              (filter_elem_num /
               skip_every); // cal output column, taking skipping into account
          if (inH + i >= 0 && inH + i < H && inW + j >= 0 && inW + j < W)
            output[((n * reduced_filter_elem + output_col) * H_out + h) *
                       W_out +
                   w] = input[((n * C + c) * H + (inH + i)) * W + (inW + j)];
          else
            output[((n * reduced_filter_elem + output_col) * H_out + h) *
                       W_out +
                   w] = 0;
        }
      }
    }
  }
}

/// This function serves as an API with the custom implementation of convolution
/// with the perforation and filter sampling support. The compute precison is FP32.
/// NOTE: This routine is used only for correctness testing
/// NOTE: This is NOT the main approximation routine used by HPVM 
void *tensorConvPerfCuda(void *input_ptr, void *filter_ptr, int vertical_pad,
                         int horizontal_pad, int vertical_stride,
                         int horizontal_stride, int conv_mode, int conv_groups,
                         int row, int col, int start) {

  Tensor *input = (Tensor *)input_ptr;
  Tensor *filter = (Tensor *)filter_ptr;
  // FIXME: Current hack to preserve backward compatibilty
  if (conv_groups == 0) {
    conv_groups = 1;
  }

  Tensor *output;
  // TODO: Support other cases;
  hostToDeviceCopy(input);
  hostToDeviceCopy(filter);

  convertToFP32(input);
  convertToFP32(filter);

  long int n, c, h, w; // output dimensions
  n = input->dims.dim_sizes[0];
  c = filter->dims.dim_sizes[0]; // number of filters
  const int KH = filter->dims.dim_sizes[2];
  const int KW = filter->dims.dim_sizes[3];

  h = (2 * vertical_pad + input->dims.dim_sizes[2] - KH) / vertical_stride + 1;
  int rem_row = (h - start) % row > 0;
  int h_eff = h - ((h - start) / row) - rem_row;

  w = (2 * horizontal_pad + input->dims.dim_sizes[3] - KW) / horizontal_stride +
      1;
  int rem_col = (w - start) % col > 0;
  int w_eff = w - ((w - start) / col) - rem_col;

  Tensor *new_output;
  if (row > 1) {
    output = (Tensor *)create4DTensor(
        (hipdnnDataType_t)float_type, // input->data_type,
        HIPDNN_TENSOR_NCHW, n, c, h_eff, w);

    // NOTE: Changing output tensor placement from host to device
    changeTensorPlacement(output, DEVICE);
    // NOTE: Necessary to insert the above call for every output tensor
    // total number of filter elem
    const long int num_filter_elem = KH * KW * input->dims.dim_sizes[1];

    float *convData;
    long int convDataSize = sizeof(float) * n * num_filter_elem * h_eff * w;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int blockSize = 128;
    const int gridSize =
        (n * input->dims.dim_sizes[1] * h_eff * w + blockSize - 1) / blockSize;

    convToGemmPerfRow<<<gridSize, blockSize>>>(
        convData, (float *)input->gpu_data, n, input->dims.dim_sizes[1],
        input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
        vertical_pad, horizontal_pad, h, w, vertical_stride, horizontal_stride,
        row, start, h_eff);
    checkCudaErrors(hipDeviceSynchronize());

    float alpha = 1.0f, beta = 0.0f;
    checkCudaErrors(hipblasSgemmStridedBatched(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h_eff * w, c, num_filter_elem,
        &alpha, convData, h_eff * w, num_filter_elem * h_eff * w,
        (float *)filter->gpu_data, num_filter_elem, 0, &beta,
        (float *)output->gpu_data, h_eff * w, c * h_eff * w, n));

    new_output = (Tensor *)create4DTensor(
        (hipdnnDataType_t)float_type, // input->data_type,
        HIPDNN_TENSOR_NCHW, n, c, h, w);
    // NOTE: Changing output tensor placement from host to device
    changeTensorPlacement(new_output, DEVICE);

    // interpolate
    int numBlocks = (n * c * h * w + 127) / 128;
    approxInterpolateRow<<<numBlocks, 128>>>(
        n * c * h * w, h_eff, n, c, h, w, (float *)output->gpu_data,
        (float *)new_output->gpu_data, row, start);
    hipDeviceSynchronize();

    freeTensor(output);
    hipFree(convData);
  } else if (col > 1) {
    output = (Tensor *)create4DTensor(
        (hipdnnDataType_t)float_type, // input->data_type,
        HIPDNN_TENSOR_NCHW, n, c, h, w_eff);

    // NOTE: Changing output tensor placement from host to device
    changeTensorPlacement(output, DEVICE);

    const long int num_filter_elem = KH * KW * input->dims.dim_sizes[1];

    float *convData;
    long int convDataSize = sizeof(float) * n * num_filter_elem * h * w_eff;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int blockSize = 128;
    const int gridSize =
        (n * input->dims.dim_sizes[1] * h * w_eff + blockSize - 1) / blockSize;

    convToGemmPerfCol<<<gridSize, blockSize>>>(
        convData, (float *)input->gpu_data, n, input->dims.dim_sizes[1],
        input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
        vertical_pad, horizontal_pad, h, w, vertical_stride, horizontal_stride,
        col, start, w_eff);
    checkCudaErrors(hipDeviceSynchronize());

    float alpha = 1.0f, beta = 0.0f;
    checkCudaErrors(hipblasSgemmStridedBatched(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h * w_eff, c, num_filter_elem,
        &alpha, convData, h * w_eff, num_filter_elem * h * w_eff,
        (float *)filter->gpu_data, num_filter_elem, 0, &beta,
        (float *)output->gpu_data, h * w_eff, c * h * w_eff, n));

    new_output = (Tensor *)create4DTensor(
        (hipdnnDataType_t)float_type, // input->data_type,
        HIPDNN_TENSOR_NCHW, n, c, h, w);
    // NOTE: Changing output tensor placement from host to device
    changeTensorPlacement(new_output, DEVICE);

    // interpolate
    int numBlocks = (n * c * h * w + 127) / 128;
    approxInterpolateCol<<<numBlocks, 128>>>(
        n * c * h * w, w_eff, n, c, h, w, (float *)output->gpu_data,
        (float *)new_output->gpu_data, col, start);
    hipDeviceSynchronize();

    freeTensor(output);
    hipFree(convData);
  } else {
    output = (Tensor *)create4DTensor(
        (hipdnnDataType_t)float_type, // input->data_type,
        HIPDNN_TENSOR_NCHW, n, c, h, w);

    // NOTE: Changing output tensor placement from host to device
    changeTensorPlacement(output, DEVICE);
    // NOTE: Necessary to insert the above call for every output tensor
    // total number of filter elem
    const long int num_filter_elem = KH * KW * input->dims.dim_sizes[1];

    float *convData;
    long int convDataSize = sizeof(float) * n * num_filter_elem * h * w;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int blockSize = 128;
    const int gridSize =
        (n * input->dims.dim_sizes[1] * h * w + blockSize - 1) / blockSize;
    convToGemmApprox<<<gridSize, blockSize>>>(
        convData, (float *)input->gpu_data, n, input->dims.dim_sizes[1],
        input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
        vertical_pad, horizontal_pad, h, w, vertical_stride, horizontal_stride,
        num_filter_elem, c * h * w);
    checkCudaErrors(hipDeviceSynchronize());

    float alpha = 1.0f, beta = 0.0f;
    checkCudaErrors(hipblasSgemmStridedBatched(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h * w, c, num_filter_elem,
        &alpha, convData, h * w, num_filter_elem * h * w,
        (float *)filter->gpu_data, num_filter_elem, 0, &beta,
        (float *)output->gpu_data, h * w, c * h * w, n));

    new_output = output;
    hipFree(convData);
  }

  return new_output;
}

__global__ void switchMatrixFull(int N, int n, int c, int h, int w,
                                 float *old_data, float *new_data) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    int col = ((i % (c * h * w)) % (h * w)) % w;
    int row = ((i % (c * h * w)) % (h * w)) / w;
    int ch = (i % (c * h * w)) / (h * w);
    int n_new = i / (c * h * w);

    new_data[((n_new * c + ch) * h + row) * w + col] =
        old_data[((ch * n + n_new) * h + row) * w + col];
  }
}

  
/*************   API for Approximation Convolution Implementations  ************/

///  ** API for FP32 Convolution that supports Baseline (No Approx), Perforation, and Filter Sampling **
/// - Arguments to control Approximation:
///    `row`: Controls the fraction of rows skipped (Perforation) - (1/row * 100)% rows skipped
///    `col`: Controls fraction of columns skipped (Perforation) - (1/col * 100)% columns skipped  
///    `skip_every`: Controls fration of filter elements skipped (Filter Sampling). (1/skip_every * 100)% filter elems skipped
///    `offset` controls the tensor index at which sampling/perforation starts
///
///   For Baseline convolution pass `row=1` `col=1` `skip_every = 1`
void *tensorConvApprox(void *input_ptr, void *filter_ptr, int vertical_pad,
                       int horizontal_pad, int vertical_stride,
                       int horizontal_stride, int conv_mode, int conv_groups,
                       int row, int col, int skip_every, int offset) {

  Tensor *input = (Tensor *)input_ptr;
  Tensor *filter = (Tensor *)filter_ptr;
  // FIXME: Current hack to preserve backward compatibilty
  if (conv_groups == 0) {
    conv_groups = 1;
  }

  hostToDeviceCopy(input);
  hostToDeviceCopy(filter);

  convertToFP32(input);
  convertToFP32(filter);

  const int n = input->dims.dim_sizes[0];
  const int c = filter->dims.dim_sizes[0]; // number of filters
  const int KH = filter->dims.dim_sizes[2];
  const int KW = filter->dims.dim_sizes[3];
  const int h = (2 * vertical_pad + input->dims.dim_sizes[2] - KH) / vertical_stride + 1;
  const int w = (2 * horizontal_pad + input->dims.dim_sizes[3] - KW) / horizontal_stride + 1;
  const int num_filter_elem = KH * KW * input->dims.dim_sizes[1];
  
  Tensor *new_output = (Tensor *)create4DTensor((hipdnnDataType_t)float_type,
                                                HIPDNN_TENSOR_NCHW, n, c, h, w);
  // NOTE: Changing output tensor placement from host to device
  changeTensorPlacement(new_output, DEVICE);
 
  if (row > 1) {
    const int rem_row = (h - offset) % row > 0;
    const int h_eff = h - ((h - offset) / row) - rem_row;

    Tensor *output = (Tensor *)create4DTensor(
        (hipdnnDataType_t)float_type, // input->data_type,
        HIPDNN_TENSOR_NCHW, n, c, h_eff, w);

    // NOTE: Changing output tensor placement from host to device
    changeTensorPlacement(output, DEVICE);

    float *convData;
    long int convDataSize = sizeof(float) * n * num_filter_elem * h_eff * w;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int blockSize = 128;
    const int gridSize =
        (n * input->dims.dim_sizes[1] * h_eff * w + blockSize - 1) / blockSize;
    convToGemmPerfRow<<<gridSize, blockSize>>>(
        convData, (float *)input->gpu_data, n, input->dims.dim_sizes[1],
        input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
        vertical_pad, horizontal_pad, h, w, vertical_stride, horizontal_stride,
        row, offset, h_eff);
    checkCudaErrors(hipDeviceSynchronize());

    float alpha = 1.0f, beta = 0.0f;
    checkCudaErrors(hipblasSgemmStridedBatched(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h_eff * w, c, num_filter_elem,
        &alpha, convData, h_eff * w, num_filter_elem * h_eff * w,
        (float *)filter->gpu_data, num_filter_elem, 0, &beta,
        (float *)output->gpu_data, h_eff * w, c * h_eff * w, n));
    // interpolate
    int blocksize = 128;
    int numBlocks = (n * c * h * w + blocksize - 1) / blocksize;
    approxInterpolateRow<<<numBlocks, blocksize>>>(
        n * c * h * w, h_eff, n, c, h, w, (float *)output->gpu_data,
        (float *)new_output->gpu_data, row, offset);
    hipDeviceSynchronize();

    freeTensor(output);
    hipFree(convData);
  } else if (col > 1) {
    const int rem_col = (w - offset) % col > 0;
    const int w_eff = w - ((w - offset) / col) - rem_col;

    Tensor *output = (Tensor *)create4DTensor(
        (hipdnnDataType_t)float_type, 
        HIPDNN_TENSOR_NCHW, n, c, h, w_eff);

    // NOTE: Changing output tensor placement from host to device
    changeTensorPlacement(output, DEVICE);

    float *convData;
    long int convDataSize = sizeof(float) * n * num_filter_elem * h * w_eff;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int blockSize = 128;
    const int gridSize =
        (n * input->dims.dim_sizes[1] * h * w_eff + blockSize - 1) / blockSize;

    convToGemmPerfCol<<<gridSize, blockSize>>>(
        convData, (float *)input->gpu_data, n, input->dims.dim_sizes[1],
        input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
        vertical_pad, horizontal_pad, h, w, vertical_stride, horizontal_stride,
        col, offset, w_eff);
    checkCudaErrors(hipDeviceSynchronize());

    float alpha = 1.0f, beta = 0.0f;
    checkCudaErrors(hipblasSgemmStridedBatched(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h * w_eff, c, num_filter_elem,
        &alpha, convData, h * w_eff, num_filter_elem * h * w_eff,
        (float *)filter->gpu_data, num_filter_elem, 0, &beta,
        (float *)output->gpu_data, h * w_eff, c * h * w_eff, n));

    // Interpolate
    int blocksize = 128;
    int numBlocks = (n * c * h * w + blocksize - 1) / blocksize;
    approxInterpolateCol<<<numBlocks, blocksize>>>(
        n * c * h * w, w_eff, n, c, h, w, (float *)output->gpu_data,
        (float *)new_output->gpu_data, col, offset);
    hipDeviceSynchronize();

    freeTensor(output);
    hipFree(convData);
  } else if (skip_every > 1) {
    // reduced number after skipping
    const int remainder = ((num_filter_elem - offset) % skip_every > 0);
    const int reduced_filter_elem =
        num_filter_elem - ((num_filter_elem - offset) / skip_every) - remainder;

    float *convData;
    size_t convDataSize = sizeof(float) * n * reduced_filter_elem * h * w;
    checkCudaErrors(hipMalloc(&convData, convDataSize));
    float *reducedFilter;
    checkCudaErrors(
        hipMalloc(&reducedFilter, sizeof(float) * c * reduced_filter_elem));

    const int filtBlockSize = 128;
    const int filtGridSize = (c * reduced_filter_elem + filtBlockSize - 1) / filtBlockSize;
    const float fac = ((float)skip_every) / ((float)skip_every - 1);
    const int blockSize = 128;
    const int gridSize = (n * h * w + blockSize - 1) / blockSize;
    
    if (!(KH * KW % skip_every)) {

      createReducedFiltersFullRegular<<<filtGridSize, filtBlockSize>>>(
          reducedFilter, (float *)filter->gpu_data, c, num_filter_elem,
          reduced_filter_elem, input->dims.dim_sizes[1], skip_every, offset,
          fac);
      checkCudaErrors(hipDeviceSynchronize());
      convToGemmFullInputRegular<<<gridSize, blockSize>>>(
          convData, (float *)input->gpu_data, n, input->dims.dim_sizes[1],
          input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
          vertical_pad, horizontal_pad, h, w, vertical_stride,
          horizontal_stride, reduced_filter_elem, skip_every, offset);
    }
    else {
      createReducedFiltersFullIrregular<<<filtGridSize, filtBlockSize>>>(
          reducedFilter, (float *)filter->gpu_data, c, num_filter_elem,
          reduced_filter_elem, skip_every, offset, fac);
      checkCudaErrors(hipDeviceSynchronize());
      convToGemmFullInputIrregular<<<gridSize, blockSize>>>(
          convData, (float *)input->gpu_data, n, input->dims.dim_sizes[1],
          input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
          vertical_pad, horizontal_pad, h, w, vertical_stride,
          horizontal_stride, reduced_filter_elem, skip_every, offset);
    }
    checkCudaErrors(hipDeviceSynchronize());

    const float alpha = 1.0;
    const float beta = 0.0;
    checkCudaErrors(hipblasSgemmStridedBatched(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h * w, c, reduced_filter_elem,
        &alpha, convData, h * w, reduced_filter_elem * h * w, reducedFilter,
        reduced_filter_elem, 0, &beta, (float *)new_output->gpu_data, h * w,
        c * h * w, n));
    hipFree(convData);
    hipFree(reducedFilter);
  } else {

    Tensor *output = (Tensor *)create4DTensor((hipdnnDataType_t)float_type,
                                              HIPDNN_TENSOR_NCHW, n, c, h, w);
    changeTensorPlacement(output, DEVICE);

    float *convData;
    long int convDataSize = sizeof(float) * n * num_filter_elem * h * w;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int blockSize = 128;
    const int gridSize =
        (n * input->dims.dim_sizes[1] * h * w + blockSize - 1) / blockSize;

    convToGemmFullInput<<<gridSize, blockSize>>>(
        convData, (float *)input->gpu_data, n, input->dims.dim_sizes[1],
        input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
        vertical_pad, horizontal_pad, h, w, vertical_stride, horizontal_stride,
        skip_every, offset); 
    
    checkCudaErrors(hipDeviceSynchronize());

    float alpha = 1.0f, beta = 0.0f;

    checkCudaErrors(hipblasGemmEx(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n * h * w, c, num_filter_elem,
        &alpha, convData, HIP_R_32F, n * h * w, (float *)filter->gpu_data,
        HIP_R_32F, num_filter_elem, &beta, (float *)output->gpu_data,
        HIP_R_32F, n * h * w, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    const int numBlocks = (n * c * h * w + 255) / 256;
    switchMatrixFull<<<numBlocks, 256>>>(n * c * h * w, n, c, h, w,
                                         (float *)output->gpu_data,
                                         (float *)new_output->gpu_data);

    checkCudaErrors(hipDeviceSynchronize());
    hipFree(convData);
  }

  return new_output;
}

__global__ void switchMatrixHalf(int N, int n, int c, int h, int w,
                                 __half *old_data, __half *new_data) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    int col = ((i % (c * h * w)) % (h * w)) % w;
    int row = ((i % (c * h * w)) % (h * w)) / w;
    int ch = (i % (c * h * w)) / (h * w);
    int n_new = i / (c * h * w);

    new_data[((n_new * c + ch) * h + row) * w + col] =
        old_data[((ch * n + n_new) * h + row) * w + col];
  }
}




///  ** API for FP16 Convolution that supports Baseline (No Approx), Perforation, and Filter Sampling **
/// - Arguments to control Approximation:
///    `row`: Controls the fraction of rows skipped (Perforation) - (1/row * 100)% rows skipped
///    `col`: Controls fraction of columns skipped (Perforation) - (1/col * 100)% columns skipped  
///    `skip_every`: Controls fration of filter elements skipped (Filter Sampling). (1/skip_every * 100)% filter elems skipped
///    `offset` controls the tensor index at which sampling/perforation starts
///
///   For Baseline convolution pass `row=1` `col=1` `skip_every = 1`
void *tensorConvApproxHalf2(void *input_ptr, void *filter_ptr, int vertical_pad,
                            int horizontal_pad, int vertical_stride,
                            int horizontal_stride, int conv_mode,
                            int conv_groups, int row, int col, int skip_every,
                            int offset) {

 
  Tensor *input = (Tensor *)input_ptr;
  Tensor *filter = (Tensor *)filter_ptr;

  // FIXME: Current hack to preserve backward compatibilty
  if (conv_groups == 0) {
    conv_groups = 1;
  }

  hostToDeviceCopy(input);
  hostToDeviceCopy(filter);

  profileEvent("F2H_start");
  convertToFP16(input);
  convertToFP16(filter);
  profileEvent("F2H_end");

  const long int n = input->dims.dim_sizes[0];
  const long int c = filter->dims.dim_sizes[0]; // number of filters
  const int KH = filter->dims.dim_sizes[2];
  const int KW = filter->dims.dim_sizes[3];
  const long int h =
      (2 * vertical_pad + input->dims.dim_sizes[2] - KH) / vertical_stride + 1;
  const long int w =
      (2 * horizontal_pad + input->dims.dim_sizes[3] - KW) / horizontal_stride +
      1;
  const long int num_filter_elem = KH * KW * input->dims.dim_sizes[1];

  Tensor *new_output = (Tensor *)create4DTensor((hipdnnDataType_t)half_type,
                                                HIPDNN_TENSOR_NCHW, n, c, h, w);
  changeTensorPlacement(new_output, DEVICE);

  const __half alf = approx_float_to_half(1.0);
  const __half bet = approx_float_to_half(0.0);
  const __half *alpha_half = &alf;
  const __half *beta_half = &bet;

  if (row > 1) {
    const int rem_row = (h - offset) % row > 0;
    const int h_eff = h - ((h - offset) / row) - rem_row;

    Tensor *output_half = (Tensor *)create4DTensor(
        (hipdnnDataType_t)half_type, HIPDNN_TENSOR_NCHW, n, c, h_eff, w);
    changeTensorPlacement(output_half, DEVICE);

    __half *convData;
    long int convDataSize = sizeof(__half) * n * num_filter_elem * h_eff * w;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int patchBlockSize = 256;
    const int numPatchBlocks =
        (n * input->dims.dim_sizes[1] * h_eff * w + patchBlockSize - 1) /
        patchBlockSize;
    const int interpolationBlocksize = 256;
    const int numInterpolationBlocks =
        (n * c * h * w + interpolationBlocksize - 1) / interpolationBlocksize;
    if (h * w <= 64) {

      convToGemmPerfRowHalf2<<<numPatchBlocks, patchBlockSize>>>(
          convData, (__half *)input->gpu_half_data, n, input->dims.dim_sizes[1],
          input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
          vertical_pad, horizontal_pad, h, w, vertical_stride,
          horizontal_stride, row, offset, h_eff);
      checkCudaErrors(hipDeviceSynchronize());

      checkCudaErrors(hipblasGemmEx(
          cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n * h_eff * w, c,
          num_filter_elem, alpha_half, convData, HIP_R_16F, n * h_eff * w,
          (__half *)filter->gpu_half_data, HIP_R_16F, num_filter_elem,
          beta_half, (__half *)output_half->gpu_half_data, HIP_R_16F,
          n * h_eff * w, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

      approxInterpolateRowHalf2<<<numInterpolationBlocks,
                                  interpolationBlocksize>>>(
          n * c * h * w, h_eff, n, c, h, w,
          (__half *)output_half->gpu_half_data,
          (__half *)new_output->gpu_half_data, row, offset);
      checkCudaErrors(hipDeviceSynchronize());

    } else {

      convToGemmPerfRowHalf<<<numPatchBlocks, patchBlockSize>>>(
          convData, (__half *)input->gpu_half_data, n, input->dims.dim_sizes[1],
          input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
          vertical_pad, horizontal_pad, h, w, vertical_stride,
          horizontal_stride, row, offset, h_eff);
      checkCudaErrors(hipDeviceSynchronize());

      checkCudaErrors(hipblasHgemmStridedBatched(
          cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h_eff * w, c, num_filter_elem,
          alpha_half, convData, h_eff * w, num_filter_elem * h_eff * w,
          (__half *)filter->gpu_half_data, num_filter_elem, 0, beta_half,
          (__half *)output_half->gpu_half_data, h_eff * w, c * h_eff * w, n));

      approxInterpolateRowHalf<<<numInterpolationBlocks,
                                 interpolationBlocksize>>>(
          n * c * h * w, h_eff, n, c, h, w,
          (__half *)output_half->gpu_half_data,
          (__half *)new_output->gpu_half_data, row, offset);
      checkCudaErrors(hipDeviceSynchronize());
    }
    freeTensor(output_half);
    hipFree(convData);
  } else if (col > 1) {
    const int rem_col = (w - offset) % col > 0;
    const int w_eff = w - ((w - offset) / col) - rem_col;

    Tensor *output_half = (Tensor *)create4DTensor(
        (hipdnnDataType_t)half_type, HIPDNN_TENSOR_NCHW, n, c, h, w_eff);
    changeTensorPlacement(output_half, DEVICE);

    __half *convData;
    long int convDataSize = sizeof(__half) * n * num_filter_elem * h * w_eff;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int patchBlockSize = 256;
    const int numPatchBlocks =
        (n * input->dims.dim_sizes[1] * h * w_eff + patchBlockSize - 1) /
        patchBlockSize;
    const int interpolationBlocksize = 256;
    const int numInterpolationBlocks =
        (n * c * h * w + interpolationBlocksize - 1) / interpolationBlocksize;
    if (h * w <= 64) {

      convToGemmPerfColHalf2<<<numPatchBlocks, patchBlockSize>>>(
          convData, (__half *)input->gpu_half_data, n, input->dims.dim_sizes[1],
          input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
          vertical_pad, horizontal_pad, h, w, vertical_stride,
          horizontal_stride, col, offset, w_eff);
      checkCudaErrors(hipDeviceSynchronize());

      checkCudaErrors(hipblasGemmEx(
          cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n * h * w_eff, c,
          num_filter_elem, alpha_half, convData, HIP_R_16F, n * h * w_eff,
          (__half *)filter->gpu_half_data, HIP_R_16F, num_filter_elem,
          beta_half, (__half *)output_half->gpu_half_data, HIP_R_16F,
          n * h * w_eff, HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

      approxInterpolateColHalf2<<<numInterpolationBlocks,
                                  interpolationBlocksize>>>(
          n * c * h * w, w_eff, n, c, h, w,
          (__half *)output_half->gpu_half_data,
          (__half *)new_output->gpu_half_data, col, offset);
      checkCudaErrors(hipDeviceSynchronize());
    }
    else {
      convToGemmPerfColHalf<<<numPatchBlocks, patchBlockSize>>>(
          convData, (__half *)input->gpu_half_data, n, input->dims.dim_sizes[1],
          input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
          vertical_pad, horizontal_pad, h, w, vertical_stride,
          horizontal_stride, col, offset, w_eff);
      checkCudaErrors(hipDeviceSynchronize());

      checkCudaErrors(hipblasHgemmStridedBatched(
          cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h * w_eff, c, num_filter_elem,
          alpha_half, convData, h * w_eff, num_filter_elem * h * w_eff,
          (__half *)filter->gpu_half_data, num_filter_elem, 0, beta_half,
          (__half *)output_half->gpu_half_data, h * w_eff, c * h * w_eff, n));

      approxInterpolateColHalf<<<numInterpolationBlocks,
                                 interpolationBlocksize>>>(
          n * c * h * w, w_eff, n, c, h, w,
          (__half *)output_half->gpu_half_data,
          (__half *)new_output->gpu_half_data, col, offset);
      checkCudaErrors(hipDeviceSynchronize());
    }

    freeTensor(output_half);
    hipFree(convData);
  } else if (skip_every > 1) {
    const int remainder = ((num_filter_elem - offset) % skip_every > 0);
    const int reduced_filter_elem =
        num_filter_elem - ((num_filter_elem - offset) / skip_every) - remainder;

    __half *convData;
    size_t convDataSize = sizeof(__half) * n * reduced_filter_elem * h * w;
    checkCudaErrors(hipMalloc(&convData, convDataSize));
    __half *reducedFilter;
    checkCudaErrors(
        hipMalloc(&reducedFilter, sizeof(__half) * c * reduced_filter_elem));

    const int filtBlockSize = 256;
    const int filtGridSize =
        (c * reduced_filter_elem + filtBlockSize - 1) / filtBlockSize;
    const float fac = ((float)skip_every) / ((float)skip_every - 1);
    const int blockSize = 256;

    const __half alf = approx_float_to_half(1.0);
    const __half bet = approx_float_to_half(0.0);
    const __half *alpha_half = &alf;
    const __half *beta_half = &bet;
    if (c * num_filter_elem <  500000) { 
      if (!(KH * KW % skip_every)) {

	createReducedFiltersHalfRegular<<<filtGridSize, filtBlockSize>>>(
            reducedFilter, (__half *)filter->gpu_half_data, c, num_filter_elem,
            reduced_filter_elem, input->dims.dim_sizes[1], skip_every, offset,
            fac);
        checkCudaErrors(hipDeviceSynchronize());

        const int gridSize =
            (n * h * w * input->dims.dim_sizes[1] + blockSize - 1) / blockSize;
        convToGemmHalfInputRegular<<<gridSize, blockSize>>>(
            convData, (__half *)input->gpu_half_data, n,
            input->dims.dim_sizes[1], input->dims.dim_sizes[2],
            input->dims.dim_sizes[3], KH, KW, vertical_pad, horizontal_pad, h,
            w, vertical_stride, horizontal_stride, reduced_filter_elem,
            skip_every, offset);
      } else {

	createReducedFiltersHalfIrregular<<<filtGridSize, filtBlockSize>>>(
            reducedFilter, (__half *)filter->gpu_half_data, c, num_filter_elem,
            reduced_filter_elem, skip_every, offset, fac);
        checkCudaErrors(hipDeviceSynchronize());

        const int gridSize =
            (n * h * w * input->dims.dim_sizes[1] + blockSize - 1) / blockSize;

	convToGemmHalfInputNewIrregular<<<gridSize, blockSize>>>(
            convData, (__half *)input->gpu_half_data, n,
            input->dims.dim_sizes[1], input->dims.dim_sizes[2],
            input->dims.dim_sizes[3], KH, KW, vertical_pad, horizontal_pad, h,
            w, vertical_stride, horizontal_stride, reduced_filter_elem,
            skip_every, offset);
      }
      checkCudaErrors(hipDeviceSynchronize());

      checkCudaErrors(hipblasHgemmStridedBatched(
          cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, h * w, c, reduced_filter_elem,
          alpha_half, convData, h * w, reduced_filter_elem * h * w,
          reducedFilter, reduced_filter_elem, 0, beta_half,
          (__half *)new_output->gpu_half_data, h * w, c * h * w, n));
    } else {
      Tensor *output_half = (Tensor *)create4DTensor(
          (hipdnnDataType_t)half_type, HIPDNN_TENSOR_NCHW, n, c, h, w);
      changeTensorPlacement(output_half, DEVICE);

      if (!(KH * KW % skip_every)) {

	createReducedFiltersHalfRegular<<<filtGridSize, filtBlockSize>>>(
            reducedFilter, (__half *)filter->gpu_half_data, c, num_filter_elem,
            reduced_filter_elem, input->dims.dim_sizes[1], skip_every, offset,
            fac);
        checkCudaErrors(hipDeviceSynchronize());

        const int gridSize =
            (n * h * w * input->dims.dim_sizes[1] + blockSize - 1) / blockSize;
        convToGemmHalfInputRegular2<<<gridSize, blockSize>>>(
            convData, (__half *)input->gpu_half_data, n,
            input->dims.dim_sizes[1], input->dims.dim_sizes[2],
            input->dims.dim_sizes[3], KH, KW, vertical_pad, horizontal_pad, h,
            w, vertical_stride, horizontal_stride, reduced_filter_elem,
            skip_every, offset);
      } else {

	createReducedFiltersHalfIrregular<<<filtGridSize, filtBlockSize>>>(
            reducedFilter, (__half *)filter->gpu_half_data, c, num_filter_elem,
            reduced_filter_elem, skip_every, offset, fac);
        checkCudaErrors(hipDeviceSynchronize());

        const int gridSize =
            (n * h * w * input->dims.dim_sizes[1] + blockSize - 1) / blockSize;
        convToGemmHalfInputNewIrregular2<<<gridSize, blockSize>>>(
            convData, (__half *)input->gpu_half_data, n,
            input->dims.dim_sizes[1], input->dims.dim_sizes[2],
            input->dims.dim_sizes[3], KH, KW, vertical_pad, horizontal_pad, h,
            w, vertical_stride, horizontal_stride, reduced_filter_elem,
            skip_every, offset);
      }
      checkCudaErrors(hipDeviceSynchronize());

      checkCudaErrors(hipblasGemmEx(
          cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n * h * w, c,
          reduced_filter_elem, alpha_half, convData, HIP_R_16F, n * h * w,
          reducedFilter, HIP_R_16F, reduced_filter_elem, beta_half,
          (__half *)output_half->gpu_half_data, HIP_R_16F, n * h * w,
          HIP_R_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

      int numBlocks = (n * c * h * w + 255) / 256;
      switchMatrixHalf<<<numBlocks, 256>>>(n * c * h * w, n, c, h, w,
                                           (__half *)output_half->gpu_half_data,
                                           (__half *)new_output->gpu_half_data);
      checkCudaErrors(hipDeviceSynchronize());

      freeTensor(output_half);
    }

    hipFree(convData);
    hipFree(reducedFilter);
  } else {

    Tensor *output = (Tensor *)create4DTensor((hipdnnDataType_t)half_type,
                                              HIPDNN_TENSOR_NCHW, n, c, h, w);

    changeTensorPlacement(output, DEVICE);
    __half *convData;
    long int convDataSize = sizeof(__half) * n * num_filter_elem * h * w;
    checkCudaErrors(hipMalloc(&convData, convDataSize));

    const int blockSize = 256;
    const int gridSize =
        (n * input->dims.dim_sizes[1] * h * w + blockSize - 1) / blockSize;

    convToGemmHalfInputNew<<<gridSize, blockSize>>>(
        convData, (__half *)input->gpu_half_data, n, input->dims.dim_sizes[1],
        input->dims.dim_sizes[2], input->dims.dim_sizes[3], KH, KW,
        vertical_pad, horizontal_pad, h, w, vertical_stride, horizontal_stride,
        num_filter_elem, skip_every, offset);
    checkCudaErrors(hipDeviceSynchronize());

    const __half alf = approx_float_to_half(1.0);
    const __half bet = approx_float_to_half(0.0);
    const __half *alpha_half = &alf;
    const __half *beta_half = &bet;
    checkCudaErrors(hipblasGemmEx(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n * h * w, c, num_filter_elem,
        alpha_half, convData, HIP_R_16F, n * h * w,
        (__half *)filter->gpu_half_data, HIP_R_16F, num_filter_elem, beta_half,
        (__half *)output->gpu_half_data, HIP_R_16F, n * h * w, HIP_R_16F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    const int numBlocks = (n * c * h * w + 255) / 256;
    switchMatrixHalf<<<numBlocks, 256>>>(n * c * h * w, n, c, h, w,
                                         (__half *)output->gpu_half_data,
                                         (__half *)new_output->gpu_half_data);
    checkCudaErrors(hipDeviceSynchronize());

    freeTensor(output);
    hipFree(convData);
  }

  profileEvent("H2F_start");
  convertToFP32_offline(new_output);

  profileEvent("H2F_end");

  return new_output;
}

} // end of Extern C
